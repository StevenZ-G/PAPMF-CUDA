#include <stdlib.h>
#include <stdio.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <time.h>

using namespace std;
#define TILE 16


/* LU Decomposition using Shared Memory \
\           CUDA                        \            
\										\	
\ ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~*/  


//Initialize a 2D matrix   
void initialize_matrices(double** a, double** l, double** u, int size)
{
  //for each row in the 2d array, initialize the values
  for (int i = 0; i < size; ++i)
  {
    a[i] = new double[size];
    l[i] = new double[size];
    u[i] = new double[size];
  }
}

__global__ void scaleIndex(double *matrix, int n, int index){
  int start=(index*n+index);
	int end=(index*n+n);
	
	for(int i= start+1 ; i<end; ++i){
		matrix[i]=(matrix[i]/matrix[start]);
	}

}

//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
// Row elimination Kernel - takes matrix, dimension, currect row index, and block size
//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
__global__ void elim(double *A, int n, int index, int bsize){
	extern __shared__ double pivot[];

	int idThread=threadIdx.x;
	int idBlock=blockIdx.x;
	int blockSize=bsize;


	if(idThread==0){
	     for(int i=index;i<n;i++) pivot[i]=A[(index*n)+i];
	}

	__syncthreads();
  //Varitables for pivot, row, start and end
	int pivotRow=(index*n);
	int currentRow=(((blockSize*idBlock) + idThread)*n);
	int start=currentRow+index;
	int end=currentRow+n;
  //If greater than pivot row, loop from start index + 1(next row) to end of column
	if(currentRow >pivotRow){
    for(int i= start+1; i<end; ++i){
        //Set the matrix value of next row and its column - pivot
        A[i]=A[i]-(A[start]*pivot[i-currentRow]);

             }
      }
}
//Randomly generated diagonal dominant (non-singular) matrix - 1D
void fillMatrix(double* a, int n){
  // Fill the matrix
   for (int i = 0; i <= (n*n); ++i) {
    a[i] =((rand()%10)+1);
  }

  //Make the matrix diagonally dominant to guarantee it is non-singular (invertible)
  int diagCount = 0;
  double sum = 0;
  for(int i = 0; i < n; ++i){
    //Iterate through the row, add all the values, remove the diagonal value
    for(int j = i*n; j < i*n + n; ++j){
      sum += abs(a[j]);
      //printf("%f +", sum);
    }
    ///Remove the diagonal value
    //i*n gives us the current row, then  add diagCount to get to correct column
    sum -= abs(a[i*n + diagCount]);
    //Add random value to the new sum, this guarantees diagonal is now larger than row sum
    a[i*n + diagCount] = sum + ((rand()%5)+1);
    ++diagCount;
    sum = 0;
  }

}

void printMatrix(double* a, int n){
    for(int i=0; i<(n*n); ++i){
           if(i%n==0)
       		   cout << endl << left << setw(9) << setprecision(3) << a[i] << left <<  setw(9);
           else cout << left << setw(9) << setprecision(3) << a[i] << left <<  setw(9);
         }
    printf("\n~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n");
    cout << endl;
}
//----------------------------------------------------------------------- 
//Print 2D Matrix
//----------------------------------------------------------------------- 
void print2D(double** matrix, int size)
{
  //for each row...
  for (int i = 0; i < size; i++)
  {
    //for each column
    for (int j = 0; j < size; j++)
    {
      //print out the cell
      cout << left << setw(9) << setprecision(3) << matrix[i][j] << left <<  setw(9);
    }
    //new line when ever row is done
    cout << endl;
  }
}

int main(int argc, char** argv){
  //Matrix dimension will be n*n
  int n = atoi(argv[1]);

  srand(1);

  //Allocate A matrix, U, and L  for CPU
  double *a = new double[n*n];
  double *ret=new double[n*n];

  //Fill in diagonal dominant matrix, then print
  fillMatrix(a, n);
 

  //Allocate GPU memory for A matrix, get number of blocks
  double *da;
  int numblock = n/TILE + ((n%TILE)?1:0);

  double runtime;
  runtime = clock()/(double)CLOCKS_PER_SEC;
  hipMalloc(&da, n*n* sizeof (double));
  hipMemcpy(da, a, n*n*sizeof(double), hipMemcpyHostToDevice);
  for(int i=0; i<n; ++i){
      scaleIndex<<<1,1>>>(da,n,i);
	    elim<<<numblock,TILE,n*sizeof(double)>>>(da,n,i,TILE);
   }
  hipMemcpy(ret, da, n*n*sizeof(double),hipMemcpyDeviceToHost );
  //printf("Returned Matrix\n");
  //printMatrix(ret,n);
  runtime = clock() - runtime;
  printf("For %u x %u Matrix\n",n,n);
  cout << "Runtime for LU Decomposition is: " << (runtime)/float(CLOCKS_PER_SEC) << endl;
  //Create 2D matrices for the L and U
  double** A = new double* [n];
  double** u = new double* [n];
  double** l = new double* [n];
 
  //Initialize these matrices, and transfer ret values into A
  initialize_matrices(A,u,l,n);
  for(int i = 0 ;i < n ; ++i){
    for(int j= 0; j < n; ++j){
      A[i][j]=ret[i*n+j];
    }
  }
  //Take values diagonal values from returned array and pull L and U
  for(int i=0; i<n; i++){
    for(int j=0; j<n; j++){
        //Find diagonals
        for(int k=0; k<n; k++){
            //If the outermost for loop is larger or equal to k, then grab L values
            if(i>=k)
              l[i][k] = A[i][k];
            //Else the rest of the array is zeroes
            else l[i][k] = 0;
            //If loops at diagonal then enter 1 for U, if j > k then we're on upper part 
            //of Matrix so fill in values, 
            if(k==j)u[k][j] = 1;
            else if(k<j)u[k][j] = A[k][j];
            else u[k][j] = 0.0;
        }
    }
  }
  //Print L and U if user specified
 if(atoi(argv[2]) == 1){
  printf("Matrix 'A' is:\n");
  printMatrix(a,n);  
  printf("~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n");
  printf("Matrix 'L' is:\n");
  print2D(l,n);
  printf("~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n");
  printf("Matrix 'U' is: \n");
  print2D(u,n);
 }


  hipFree(da);
  hipFree(ret);
  delete[] a;
  delete[] ret; 

  return 0;
}