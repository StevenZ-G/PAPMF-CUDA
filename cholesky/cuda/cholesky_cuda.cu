#include <stdlib.h>
#include <stdio.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <time.h>

using namespace std;
#define TILE 16

  
// Inicializa las matrices en memoria del dispositivo (GPU)
void initialize_matrices(double** da, double** dl, int size) {
    // Asignar memoria en el dispositivo para las matrices A y L
    hipMalloc((void**)da, size * size * sizeof(double));  // para la matriz A
    hipMalloc((void**)dl, size * size * sizeof(double));  // para la matriz L
    
    // Inicializa la matriz L con ceros
    hipMemset(*dl, 0, size * size * sizeof(double));
}

__global__ void scaleIndex(double *matrix, int n, int index){
  int start=(index*n+index);
	int end=(index*n+n);
	
	for(int i= start+1 ; i<end; ++i){
		matrix[i]=(matrix[i]/matrix[start]);
	}

}

__global__ void elim(double *A, int n, int index, int bsize){
	extern __shared__ double pivot[];

	int idThread=threadIdx.x;
	int idBlock=blockIdx.x;
	int blockSize=bsize;


	if(idThread==0){
	     for(int i=index;i<n;i++) pivot[i]=A[(index*n)+i];
	}

	__syncthreads();
  //Varitables for pivot, row, start and end
	int pivotRow=(index*n);
	int currentRow=(((blockSize*idBlock) + idThread)*n);
	int start=currentRow+index;
	int end=currentRow+n;
  //If greater than pivot row, loop from start index + 1(next row) to end of column
	if(currentRow >pivotRow){
    for(int i= start+1; i<end; ++i){
        //Set the matrix value of next row and its column - pivot
        A[i]=A[i]-(A[start]*pivot[i-currentRow]);

             }
      }
}
//Randomly generated diagonal dominant (non-singular) matrix - 1D
void fillMatrix(double* a, int n){
  // Fill the matrix
   for (int i = 0; i <= (n*n); ++i) {
    a[i] =((rand()%10)+1);
  }

  //Make the matrix diagonally dominant to guarantee it is non-singular (invertible)
  int diagCount = 0;
  double sum = 0;
  for(int i = 0; i < n; ++i){
    //Iterate through the row, add all the values, remove the diagonal value
    for(int j = i*n; j < i*n + n; ++j){
      sum += abs(a[j]);
      //printf("%f +", sum);
    }
    ///Remove the diagonal value
    //i*n gives us the current row, then  add diagCount to get to correct column
    sum -= abs(a[i*n + diagCount]);
    //Add random value to the new sum, this guarantees diagonal is now larger than row sum
    a[i*n + diagCount] = sum + ((rand()%5)+1);
    ++diagCount;
    sum = 0;
  }

}

void printMatrix(double* a, int n){
    for(int i=0; i<(n*n); ++i){
           if(i%n==0)
       		   cout << endl << left << setw(9) << setprecision(3) << a[i] << left <<  setw(9);
           else cout << left << setw(9) << setprecision(3) << a[i] << left <<  setw(9);
         }
    printf("\n~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n");
    cout << endl;
}
//----------------------------------------------------------------------- 
//Print 2D Matrix
//----------------------------------------------------------------------- 
void print2D(double** matrix, int size)
{
  //for each row...
  for (int i = 0; i < size; i++)
  {
    //for each column
    for (int j = 0; j < size; j++)
    {
      //print out the cell
      cout << left << setw(9) << setprecision(3) << matrix[i][j] << left <<  setw(9);
    }
    //new line when ever row is done
    cout << endl;
  }
}

__global__ void choleskyScale(double* da, int n, int i) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) return;

    if (index == i) {
        // Diagonal element: compute sqrt
        da[i * n + i] = sqrt(da[i * n + i]);
    }
}

__global__ void choleskyElim(double* da, int n, int i, int tile) {
    int row = blockIdx.x * tile + threadIdx.x;
    int col = blockIdx.y * tile + threadIdx.y;

    if (row >= n || col >= n) return;

    if (row > i && col >= i) {
        // Eliminate elements using Cholesky formula: L[i,j] = (A[i,j] - sum) / L[i,i]
        double sum = 0.0;
        for (int k = 0; k < i; ++k) {
            sum += da[row * n + k] * da[col * n + k];
        }
        da[row * n + col] = (da[row * n + col] - sum) / da[i * n + i];
    }
}


int main(int argc, char** argv) {
    // Definimos la dimensión de la matriz
    int n = atoi(argv[1]);

    srand(1);

    // Asignamos memoria para la matriz A y el resultado (matriz L)
    double* a = new double[n * n];
    double* ret = new double[n * n];

    // Llenamos la matriz A
    fillMatrix(a, n);

    // Asignamos memoria en el dispositivo para A y L
    double* da;
    double* dl;
    int numblock = n / TILE + ((n % TILE) ? 1 : 0);

    double runtime;
    runtime = clock() / (double)CLOCKS_PER_SEC;
    
    // Asignamos memoria para las matrices en el dispositivo
    initialize_matrices(&da, &dl, n);

    // Transferimos la matriz A del host al dispositivo
    hipMemcpy(da, a, n * n * sizeof(double), hipMemcpyHostToDevice);

    // Descomposición de Cholesky: Calculamos L
    for (int i = 0; i < n; ++i) {
        // Escalamos el valor diagonal
        choleskyScale<<<1, 1>>>(da, n, i);

        // Eliminamos los valores debajo de la diagonal
        choleskyElim<<<dim3(numblock, numblock), dim3(TILE, TILE)>>>(da, n, i, TILE);
    }

    // Transferimos el resultado de vuelta al host
    hipMemcpy(ret, da, n * n * sizeof(double), hipMemcpyDeviceToHost);

    runtime = clock() - runtime;
    printf("For %u x %u Matrix\n", n, n);
    std::cout << "Runtime for Cholesky Decomposition is: " << (runtime) / float(CLOCKS_PER_SEC) << std::endl;

    // Crear matrices 2D para L
    double** l = new double* [n];
    
    // La inicialización de la matriz L debe realizarse en el host, no en la GPU
    // Asignamos memoria para la matriz L en el host
    for (int i = 0; i < n; ++i) {
        l[i] = new double[n];
    }

    // Extraemos L del resultado
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i >= j) {
                l[i][j] = ret[i * n + j];
            } else {
                l[i][j] = 0.0;
            }
        }
    }

    // Imprimir matriz L
    if (atoi(argv[2]) == 1) {
        printf("Matrix 'A' is:\n");
        printMatrix(a, n);
        printf("~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n");
        printf("Matrix 'L' is:\n");
        print2D(l, n);
    }

    // Liberamos memoria
    hipFree(da);
    hipFree(dl);
    delete[] a;
    delete[] ret;

    return 0;
}
